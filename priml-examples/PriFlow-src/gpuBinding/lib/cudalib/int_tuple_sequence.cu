#include "hip/hip_runtime.h"
#include "../headers/export.h"
#include "../headers/hofs.h"

//Builtin function pointers
#include "../funcptrs/builtin_filter_int_tuple.h"
#include "../funcptrs/builtin_reduce_and_scan_int_tuple.h"
#include "../funcptrs/builtin_tabulate_and_map_int_tuple.h"

#include "../funcptrs/user_map_int_tuple.h"
#include "../funcptrs/user_reduce_int_tuple.h"
#include "../funcptrs/user_scan_int_tuple.h"
#include "../funcptrs/user_filter_int_tuple.h"
#include "../funcptrs/user_zipwith_int_tuple.h"

#include <stdio.h>
#include <time.h>
#include <utility>

#define blockSize = 256

#define threads_reduce 1024
#define block_red_size_reduce (threads_reduce / 32)

#define threads_scan 1024
#define block_red_size_scan (threads_scan / 32)

#define threads_filter 256

//Tabulate
__global__ 
void tabulate_int_tuple_kernel(int* arr_1, int* arr_2, int len, tabulate_fun_int_tuple f){
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= len){
    return;
  }

  int2 T = f(idx);
  arr_1[idx] = T.x;
  arr_2[idx] = T.y;
}

extern "C"
void tabulate_int_tuple(int size, void* f, Pointer dev_ptr_1, Pointer dev_ptr_2){
  
  tabulate_fun_int_tuple hof = (tabulate_fun_int_tuple)f;
  
  hipMalloc((void**)dev_ptr_1, sizeof(int) * size);
  hipMalloc((void**)dev_ptr_2, sizeof(int) * size);

  int blockNum = (size / 256) + 1;
  tabulate_int_tuple_kernel<<<blockNum, 256>>>(*(int**)dev_ptr_1, *(int**)dev_ptr_2, size, hof);
  hipDeviceSynchronize();

}

//Map
__global__
void map_int_tuple_kernel(int* arr_1, int* arr_2, int len, map_fun_int_tuple f){
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= len){
    return;
  }

  int2 tuple = f(arr_1[idx],arr_2[idx]);
  arr_1[idx] = tuple.x;
  arr_2[idx] = tuple.y;
}

extern "C"
void map_int_tuple(void* inarr_1, void* inarr_2, void* f, int size){
  
  map_fun_int_tuple hof = (map_fun_int_tuple)f;      
  int blockNum = (size / 256) + 1;
  
  map_int_tuple_kernel<<<blockNum, 256>>>((int*)inarr_1, (int*)inarr_2, size, hof);
}


__inline__ __device__
int2 warp_red_int_tuple(int t_1, int t_2, reduce_fun_int_tuple f){
  int res_1 = t_1;
  int res_2 = t_2;
  int2 res;
  #pragma unroll
  for(int i = 16;i > 0;i /= 2){
    int a1 = __shfl_down(res_1, i);
    int a2 = __shfl_down(res_2, i);
    res = f(res_1, res_2, a1, a2);
    res_1 = res.x;
    res_2 = res.y;

  }
  return res;
}

__inline__ __device__
int2 reduce_block_int_tuple(int t_1, int t_2, int b_1, int b_2, reduce_fun_int_tuple f){
  
  // assuming warp size is 32
  // can fix later in the kernel call
  __shared__ int warp_reds_1[block_red_size_reduce];
  __shared__ int warp_reds_2[block_red_size_reduce];

  int warpIdx = threadIdx.x / warpSize;

  int localIdx = threadIdx.x % warpSize;

  int2 inter_res = warp_red_int_tuple(t_1, t_2, f);
  
  if(localIdx == 0){
    warp_reds_1[warpIdx] = inter_res.x;
    warp_reds_2[warpIdx] = inter_res.y;
  }

  __syncthreads();
  
  int broadval2_1 = (threadIdx.x < block_red_size_reduce) ? warp_reds_1[localIdx] : b_1;
  int broadval2_2 = (threadIdx.x < block_red_size_reduce) ? warp_reds_2[localIdx] : b_2;

  int2 res = make_int2(b_1, b_2);
  if(warpIdx == 0){
    res = warp_red_int_tuple(broadval2_1, broadval2_2, f);
  }

  return res;
}

__global__
void reduce_int_tuple_kernel(int* in_1, int* in_2, int* out_1, int* out_2, int size, int b_1, int b_2, reduce_fun_int_tuple f){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int sum_1 = b_1;
  int sum_2 = b_2;
  int2 sum;
  
  #pragma unroll
  for(int i = idx; i < size; i += blockDim.x * gridDim.x){
    sum = f(sum_1, sum_2, in_1[i], in_2[i]);
    sum_1 = sum.x;
    sum_2 = sum.y;
  }
  
  sum = reduce_block_int_tuple(sum_1, sum_2, b_1, b_2, f);
  
  if(threadIdx.x == 0){
    out_1[blockIdx.x] = sum.x;
    out_2[blockIdx.x] = sum.y;
  }
  
}

// cite : https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler
// for algorithm / ideas on how to use shfl methods for fast reductions
extern "C"
void reduce_int_tuple_shfl(void* arr_1, void* arr_2, int size, int b_1, int b_2, void* f, Pointer out_1, Pointer out_2){

  reduce_fun_int_tuple hof = (reduce_fun_int_tuple) f;
  

  int numBlocks = (size / threads_reduce) + 1;
  void* res_1;
  void* res_2;

  hipMalloc(&res_1, sizeof(int) * numBlocks);
  hipMalloc(&res_2, sizeof(int) * numBlocks);
  reduce_int_tuple_kernel<<<numBlocks, threads_reduce>>>
          ((int*)arr_1, (int*)arr_2, (int*)res_1, (int*)res_2, 
                                                   size, b_1, b_2, hof);
  reduce_int_tuple_kernel<<<1, 1024>>>
          ((int*)res_1, (int*)res_2, (int*)res_1, (int*)res_2, numBlocks, b_1, b_2, hof);

  hipMemcpy(&out_1, res_1, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&out_2, res_2, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(res_1);
  hipFree(res_2);
}

//BEGIN SCAN

__device__ __inline__
int2 warp_scan_shfl(int b_1, int b_2, scan_fun_int_tuple f, int* out_1, int* out_2, int idx, int length){
  int warpIdx = threadIdx.x % warpSize;
  int2 res;
  if(idx < length){
    res.x = out_1[idx];
    res.y = out_2[idx];
  }
  else{
    res.x = b_1;
    res.y = b_2;
  }
  #pragma unroll
  for(int i = 1;i < warpSize;i *= 2){
    int a_1 = __shfl_up(res.x, i);
    int a_2 = __shfl_up(res.y, i);
    if(i <= warpIdx){
      res = f(a_1, a_2, res.x, res.y);
    }
  }
  if(idx < length){
    out_1[idx] = res.x;
    out_2[idx] = res.y;
  }
  return res;
}

__device__ __inline__
int2 block_scan(int* in_1, int* in_2, int length, scan_fun_int_tuple f, int b_1, int b_2){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ int warp_reds_1[block_red_size_scan];
  __shared__ int warp_reds_2[block_red_size_scan];

  int warpIdx = threadIdx.x / warpSize;

  int localIdx= threadIdx.x % warpSize;

  int2 inter_res = warp_scan_shfl(b_1, b_2, f, in_1, in_2, idx, length);

  if(localIdx == warpSize - 1){
    warp_reds_1[warpIdx] = inter_res.x;
    warp_reds_2[warpIdx] = inter_res.y;
  }

  __syncthreads();

  int2 res = make_int2(b_1, b_2);
  if(warpIdx == 0){
    res = warp_scan_shfl(b_1, b_2, f, warp_reds_1, warp_reds_2, localIdx, block_red_size_scan);
  }
  
  __syncthreads();

  if(idx < length && warpIdx != 0){
    int2 t = 
        f(warp_reds_1[warpIdx - 1], warp_reds_2[warpIdx - 1], in_1[idx], in_2[idx]);
    in_1[idx] = t.x;
    in_2[idx] = t.y;
  }

  //warp number 0, lane number block_red_size_scan 
  //will return the final result for scanning over this
  //block 
  return res;
}

//inclusive kernel
__global__
void scan_int_tuple_kernel(int* in_1, int* in_2, int* block_results1, int* block_results2, 
                          scan_fun_int_tuple f, int b1, int b2, int length){
  
  int2 block_res = block_scan(in_1, in_2, length, f, b1, b2);
  if(threadIdx.x == block_red_size_scan - 1){
    block_results1[blockIdx.x] = block_res.x;
    block_results2[blockIdx.x] = block_res.y;
  }
}
__global__
void compress_results(int* block_res1, int* block_res2, int* out1, int* out2, 
                      int len, scan_fun_int_tuple f){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(blockIdx.x == 0){
    return;
  }
  else{
    if(idx < len){
      int2 t = 
        f(block_res1[blockIdx.x - 1], block_res2[blockIdx.x - 1], out1[idx], out2[idx]);
      out1[idx] = t.x;
      out2[idx] = t.y;
    }
  }
}

//this is terrible
__global__
void serial_scan(int* bres_1, int* bres_2, int len, int b_1, int b_2, scan_fun_int_tuple f){
  int2 res = make_int2(b_1, b_2);
  #pragma unroll
  for(int i = 0; i < len; i++){
    res = f(res.x, res.y, bres_1[i], bres_2[i]);
    bres_1[i] = res.x;
    bres_2[i] = res.y;
  }
}

extern "C"
void inclusive_scan_int_tuple(void* in_1, void* in_2, void* f, int length, int b_1, int b_2){
  
  scan_fun_int_tuple hof = (scan_fun_int_tuple)f;

  int num_blocks_first = (length / threads_scan) + 1;
  int* block_results_1;
  int* block_results_2;
  int* dummy_1;
  int* dummy_2;
  hipMalloc(&block_results_1, sizeof(int) * num_blocks_first);
  hipMalloc(&block_results_2, sizeof(int) * num_blocks_first);
  hipMalloc(&dummy_1, sizeof(int));
  hipMalloc(&dummy_2, sizeof(int));

  scan_int_tuple_kernel<<<num_blocks_first, threads_scan>>>
      ((int*)in_1, (int*)in_2, block_results_1, block_results_2, hof, b_1, b_2, length);

  if(num_blocks_first == 1){
    hipDeviceSynchronize();
    hipFree(block_results_1);
    hipFree(block_results_2);
    hipFree(dummy_1);
    hipFree(dummy_2);
    return;
  }
  else if(num_blocks_first <= 1024){
    scan_int_tuple_kernel<<<1, 1024>>>
            (block_results_1, block_results_2, dummy_1, dummy_2, hof, b_1, b_2, num_blocks_first);
    compress_results<<<num_blocks_first, threads_scan>>>
            (block_results_1, block_results_2, (int*)in_1, (int*)in_2, length, hof);
    hipDeviceSynchronize();
    hipFree(block_results_1);
    hipFree(block_results_2);
    hipFree(dummy_1);
    hipFree(dummy_2);
    return;
  }
  else{
    int leftover = (num_blocks_first / threads_scan) + 1;
    int* block_block_results_1;
    int* block_block_results_2;
    hipMalloc(&block_block_results_1, sizeof(int) * leftover);
    hipMalloc(&block_block_results_2, sizeof(int) * leftover);

    scan_int_tuple_kernel<<<leftover, threads_scan>>>
            (block_results_1, block_results_2, block_block_results_1, 
             block_results_2, hof, b_1, b_2, num_blocks_first);

    serial_scan<<<1,1>>>(block_block_results_1, block_block_results_2, leftover, b_1, b_2, hof);

    compress_results<<<leftover, threads_scan>>>
      (block_block_results_1, block_block_results_2, block_results_1, 
       block_results_2, num_blocks_first, hof);
    compress_results<<<num_blocks_first, threads_scan>>>
      (block_results_1, block_results_2, (int*)in_1, (int*)in_2, length, hof);

    hipDeviceSynchronize();
    hipFree(block_results_1);
    hipFree(block_results_2);
    hipFree(dummy_1);
    hipFree(dummy_2);
    hipFree(block_block_results_1);
    hipFree(block_block_results_2);
    return;
  }
}

//BEGIN EXCLUSIVE SCAN

__global__
void excl_compress_results(int* block_res_1, int* block_res_2, int* out_1, int* out_2, int len, scan_fun_int_tuple f, int* final, int b_1, int b_2){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx >= len) return;
  if(blockIdx.x != 0){
    int2 t = f(block_res_1[blockIdx.x - 1], block_res_2[blockIdx.x - 1],
                            out_1[idx], out_2[idx]);
    out_1[idx] = t.x;
    out_2[idx] = t.y;
  }
  __syncthreads();
  int2 toWrite = make_int2(b_1, b_2);
  if(threadIdx.x == 0){
    if(idx == 0){
      toWrite.x = b_1;
      toWrite.y = b_2;
    }
    else{
      toWrite.x = block_res_1[blockIdx.x - 1];
      toWrite.y = block_res_2[blockIdx.x - 1];
    }
  }
  else{
    toWrite.x = out_1[idx - 1];
    toWrite.y = out_2[idx - 1];
  }
  if(idx == len - 1){
    final[0] = out_1[idx];
    final[1] = out_2[idx];
  }
  __syncthreads();
  out_1[idx] = toWrite.x;
  out_2[idx] = toWrite.y;
}

extern "C"
void exclusive_scan_int_tuple(void* in_1, void* in_2, void* f, int length, int b_1, int b_2, Pointer out_1, Pointer out_2){
  
  scan_fun_int_tuple hof = (scan_fun_int_tuple)f;

  int num_blocks_first = (length / threads_scan) + 1;
  int* block_results_1, *block_results_2;
  int* dummy_1, *dummy_2;
  int* final_val;
  hipMalloc(&block_results_1, sizeof(int) * num_blocks_first);
  hipMalloc(&block_results_2, sizeof(int) * num_blocks_first);
  hipMalloc(&dummy_1, sizeof(int));
  hipMalloc(&dummy_2, sizeof(int));
  hipMalloc(&final_val, 2 * sizeof(int));


  scan_int_tuple_kernel<<<num_blocks_first, threads_scan>>>
          ((int*)in_1, (int*)in_2, block_results_1, block_results_2, hof, b_1, b_2, length);
  int2 res;
  if(num_blocks_first == 1){
    excl_compress_results<<<num_blocks_first, threads_scan>>>
          (block_results_1, block_results_2, (int*)in_1, (int*)in_2, length, hof, final_val, b_1, b_2);
    hipMemcpy(&res.x, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&res.y, final_val+1, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results_1);
    hipFree(block_results_2);
    hipFree(dummy_1);
    hipFree(dummy_2);
    hipFree(final_val);
    *(int*)out_1 = res.x;
    *(int*)out_2 = res.y;
  }
  else if(num_blocks_first <= 1024){
    scan_int_tuple_kernel<<<1, 1024>>>(block_results_1, block_results_2, dummy_1, dummy_2, hof, b_1, b_2, num_blocks_first);
    excl_compress_results<<<num_blocks_first, threads_scan>>>
            (block_results_1, block_results_2, (int*)in_1, (int*)in_2, 
             length, hof, final_val, b_1, b_2);
    hipMemcpy(&res.x, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&res.y, final_val+1, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results_1);
    hipFree(block_results_2);
    hipFree(dummy_1);
    hipFree(dummy_2);
    hipFree(final_val);
    *(int*)out_1 = res.x;
    *(int*)out_2 = res.y;
  }
  else{
    int leftover = (num_blocks_first / threads_scan) + 1;
    int* block_block_results_1, *block_block_results_2;
    hipMalloc(&block_block_results_1, sizeof(int) * leftover);
    hipMalloc(&block_block_results_2, sizeof(int) * leftover);
    scan_int_tuple_kernel<<<leftover, threads_scan>>>
            (block_results_1, block_results_2, block_block_results_1, block_block_results_1, hof, b_1, b_2, num_blocks_first);
    serial_scan<<<1,1>>>(block_block_results_1, block_block_results_2, leftover, b_1, b_2, hof);
    compress_results<<<leftover, threads_scan>>>
            (block_block_results_1, block_block_results_2, block_results_1, block_results_2 , num_blocks_first, hof);
    excl_compress_results<<<num_blocks_first, threads_scan>>>
            (block_results_1, block_results_2, (int*)in_1, (int*)in_2, length, hof, final_val, b_1, b_2);
    hipMemcpy(&res, final_val, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(block_results_1);
    hipFree(block_results_2);
    hipFree(dummy_1);
    hipFree(dummy_2);
    hipFree(final_val);
    hipFree(block_block_results_1);
    hipFree(block_block_results_2);
    *(int*)out_1 = res.x;
    *(int*)out_2 = res.y;
  }
}

__global__
void filter_map(int* in_1, int* in_2, int* out1, int len, filter_fun_int_tuple f){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < len){
    if(f(in_1[idx],in_2[idx])){
      out1[idx] = 1;
    }
    else{
      out1[idx] = 0;
    }
  }
}
__global__
void squish(int* in_1, int* in_2, int* scanned, int* out_1, int* out_2, int length, filter_fun_int_tuple f){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if(idx < length){
    if(f(in_1[idx], in_2[idx]) == 1){
      out_1[scanned[idx]] = in_1[idx];
      out_2[scanned[idx]] = in_2[idx];
    }
  }
}

__device__
int add_int_(int x, int y){
  return x+y;
}
__device__ reduce_fun_int add_devl_int = add_int_;
/*
extern "C"
void* filter_int_tuple(void* arr_1, void* arr_2, int length, void* f, Pointer out_1, Pointer out_2, Pointer outlen){
  filter_fun_int_tuple hof = (filter_fun_int_tuple)f;
  
  int blocks = (length / threads_filter) + 1;
    
  // make buffer array

  // this map could have been fused in with the scan with some 
  // extra code copy pasta I didn't want to do

  int* scanned;
  hipMalloc(&scanned, sizeof(int) * length);
  filter_map<<<blocks, threads_filter>>>((int*)arr_1, (int*)arr_2, scanned, length, hof);
  
  //scan over the bits
  reduce_fun_int add;
  hipMemcpyFromSymbol(&add, HIP_SYMBOL(add_devl_int), sizeof(reduce_fun_int));
  int len = exclusive_scan_int(scanned, (void*)add, length, 0);

  hipMalloc((int*)out_1, sizeof(int) * len);
  hipMalloc((int*)out_2, sizeof(int) * len);

  squish<<<blocks, threads_filter>>>((int*)arr_1, (int*)arr_2, scanned, out_1, out_2, length, hof);
  *(int*)outlen = len;
  hipFree(scanned);
}


__global__
<<<<<<< HEAD
void zipsquish(int* arr1, int* arr2, int* out, zipwith_fun_int_tuple f, int length){
=======
void zipsquish(int* arr1_1, int* arr2_1, int* arr1_2, int* arr2_2, int* out_1, int* out_2, zipwith_fun_int_tuple f, int length){
>>>>>>> 4e775e2bfec8033ee7238553b36a9b854b81fc01
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  int2 out;
  if(idx < length){
    out = f(arr1_1[idx], arr2_1[idx], arr1_2[idx], arr2_2[idx]);
    out_1[idx] = out.x;
    out_2[idx] = out.y;
  }
}

extern "C"
void zipwith_int_tuple(int* arr1_1, int* arr2_1, int* arr1_2, int* arr2_2, void* f, int length, Pointer out_1, Pointer out_2){

  zipwith_fun_int_tuple hof = (zipwith_fun_int_tuple)f;
  
  hipMalloc((int*)out_1, sizeof(int) * len);
  hipMalloc((int*)out_2, sizeof(int) * len);

  int blocks = (length / threads_filter) + 1;
  zipsquish<<<blocks, threads_filter>>>((int*)arr1_1, (int*)arr2_1, (int*)arr1_2, (int*)arr2_2, (int*)out_1, (int*)out_2, hof, length);

  hipDeviceSynchronize();
}
*/
//Reduce - cite http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf - another reduction algorithm choice
